#include "hip/hip_runtime.h"


//#include "cuda_helper.cuh"
#include "cuda_header.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <math.h>




//
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>

texture<float, 3, hipReadModeElementType> texPtr;

__global__ void test_kernel(void) {
	printf("Hello, world! from GPU\n");
}


__global__ void texture_kernel(float *dst, int width, int height)
{
	unsigned int x = threadIdx.x;
	unsigned int y = threadIdx.y;
	unsigned int z = blockIdx.x;

	float sample = tex3D(texPtr, x, y, z);

	dst[z * width * height + y * width + x] = sample;

}


void wrapper(void)
{
	test_kernel << <1, 1 >> > ();
	printf("Hello, world!\n");
	hipDeviceSynchronize();
}

void texture_test(void)
{
	int width = 2;
	int height = 2;
	int depth = 2;
	float *src = (float*)malloc(sizeof(float)*width*height*depth);
	float *dst = (float*)malloc(sizeof(float)*width*height*depth);
	float* src_d;
	float* dst_d;

	hipError_t result;
	hipArray * cu_array;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent extent;
	extent.width = width;
	extent.height = height;
	extent.depth = depth;

	result = hipMalloc((void**)&dst_d, sizeof(float)*width*height*depth);
	result = hipMalloc3DArray(&cu_array, &channelDesc, extent, 0);
	if (result != hipSuccess) {
		fprintf(stderr, "Texture3D - failed to malloc 3D array - %s \n", hipGetErrorString(result));
		return;
	}

	for (int i = 0; i < width*height*depth; i++) {
		src[i] = i;
	}

	hipMemcpy3DParms params;

	memset(&params, 0, sizeof(params));
	params.srcPtr.pitch = sizeof(float)* width;
	params.srcPtr.ptr = src;
	params.srcPtr.xsize = width;
	params.srcPtr.ysize = height;

	params.srcPos.x = 0;
	params.srcPos.y = 0;
	params.srcPos.z = 0;

	params.dstArray = cu_array;

	params.dstPos.x = 0;
	params.dstPos.y = 0;
	params.dstPos.z = 0;

	params.extent.width = width;
	params.extent.depth = depth;
	params.extent.height = height;

	params.kind = hipMemcpyHostToDevice;

	result = hipMemcpy3D(&params);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from host buffer to device array - %s\n", hipGetErrorString(result));
		return;
	}

	texPtr.addressMode[0] = hipAddressModeWrap;
	texPtr.addressMode[1] = hipAddressModeWrap;
	texPtr.addressMode[2] = hipAddressModeWrap;
	texPtr.filterMode = hipFilterModePoint;// hipFilterModePoint or hipFilterModeLinear
	texPtr.normalized = false;

	// bind to array
	result = hipBindTextureToArray(texPtr, cu_array, channelDesc);
	if (result != hipSuccess) {
		fprintf(stderr, "hipBindTextureToArray() - failed to bind texture to array - %s", hipGetErrorString(result));
		return;
	}

	texture_kernel << <dim3(depth, 1), dim3(width, height) >> > (dst_d, width, height);

	hipDeviceSynchronize();
	result = hipMemcpy(dst, dst_d, sizeof(float)*width*height*depth, hipMemcpyDeviceToHost);

	for (int i = 0; i < width*height*depth; i++) {
		printf("%d %.1f == %.1f \n", i, src[i], dst[i]);
	}printf("\n\n");

	// Texture �޸� ���� �����ϱ�
	float *srcSingleChannel = (float*)malloc(sizeof(float)*width*height);
	for (int i = 0; i < width*height; i++) {
		srcSingleChannel[i] = 100 + i;
	}

	params.srcPtr.ptr = srcSingleChannel;
	params.dstPos.z = 1;
	params.extent.depth = 1;
	result = hipMemcpy3D(&params);

	texture_kernel << <dim3(depth, 1), dim3(width, height) >> > (dst_d, width, height);

	hipDeviceSynchronize();
	result = hipMemcpy(dst, dst_d, sizeof(float)*width*height*depth, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from host buffer to device array - %s\n", hipGetErrorString(result));
		return;
	}

	for (int i = 0; i < width*height*depth; i++) {
		printf("%d %.1f == %.1f \n", i, src[i], dst[i]);
	}
	hipDeviceSynchronize();


}



//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
